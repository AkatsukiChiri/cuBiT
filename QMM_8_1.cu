#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "/home/mashaobo/cuBiT/tools/common.cuh"

__global__ void test(uint *A, uint *B){
    printf("%x\n",*A);
    printf("%x\n",*B);
    printf("%x\n", __popc(*A));
    printf("%x\n", __popc(*B));
    return;
}

int main(){
    uint A = 0xabcdabcd;
    uint B = 0x0f0f0f0f;
    
    uint *dA, *dB;

    printf("cpu%x\n",A);
    printf("cpu%x\n",B);

    hipMalloc(&dA, sizeof(uint));
    hipMalloc(&dB, sizeof(uint));
    
    hipMemcpy(dA, &A, sizeof(uint), hipMemcpyHostToDevice); 
    hipMemcpy(dB, &B, sizeof(uint), hipMemcpyHostToDevice); 

    test<<<1,1>>>(dA, dB);
    hipDeviceSynchronize();

    hipFree(dA);
    hipFree(dB);
    return 0;
}
